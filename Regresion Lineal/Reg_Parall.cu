#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>

// Estructura para almacenar los resultados de la regresión
struct LinearRegressionResult {
    double slope;      // Pendiente (m)
    double intercept;  // Intersección (b)
};

// Función para predecir el valor con un modelo de regresión lineal
_host_ _device_ double predict(const double& feature, const LinearRegressionResult& result) {
    return result.slope * feature + result.intercept;
}

// Kernel para el entrenamiento del modelo en paralelo con CUDA
_global_ void trainModelKernel(const double* X, const double* y, size_t numSamples,
                                 int epochs, double learningRate, LinearRegressionResult* result) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numSamples) {
        double slope = result->slope;
        double intercept = result->intercept;

        for (int epoch = 0; epoch < epochs; ++epoch) {
            // Predicción
            double y_pred = predict(X[idx], {slope, intercept});

            // Cálculo del error
            double error = y_pred - y[idx];

            // Actualización de pesos
            slope -= learningRate * error * X[idx];
            intercept -= learningRate * error;
        }

        // Almacenar resultados finales en el resultado compartido
        result->slope = slope;
        result->intercept = intercept;
    }
}

// Función para entrenar el modelo utilizando CUDA
LinearRegressionResult trainModelCUDA(const std::vector<double>& X, const std::vector<double>& y,
                                       int epochs, double learningRate) {
    size_t numSamples = X.size();

    // Copiar datos desde la CPU a la GPU
    double* d_X;
    double* d_y;
    hipMalloc((void**)&d_X, numSamples * sizeof(double));
    hipMalloc((void**)&d_y, numSamples * sizeof(double));
    hipMemcpy(d_X, X.data(), numSamples * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), numSamples * sizeof(double), hipMemcpyHostToDevice);

    // Inicializar resultado en la GPU
    LinearRegressionResult result;
    result.slope = 0.0;
    result.intercept = 0.0;

    LinearRegressionResult* d_result;
    hipMalloc((void**)&d_result, sizeof(LinearRegressionResult));
    hipMemcpy(d_result, &result, sizeof(LinearRegressionResult), hipMemcpyHostToDevice);

    // Configurar el tamaño del bloque y la cuadrícula
    size_t blockSize = 256;
    size_t gridSize = (numSamples + blockSize - 1) / blockSize;

    // Ejecutar el kernel en la GPU
    trainModelKernel<<<gridSize, blockSize>>>(d_X, d_y, numSamples, epochs, learningRate, d_result);

    // Copiar resultados de la GPU a la CPU
    hipMemcpy(&result, d_result, sizeof(LinearRegressionResult), hipMemcpyDeviceToHost);

    // Liberar memoria en la GPU
    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_result);

    return result;
}

int main() {
    auto start = std::chrono::high_resolution_clock::now();

    std::ifstream inputFile("input.txt");
    if (!inputFile.is_open()) {
        std::cerr << "Error al abrir el archivo de entrada." << std::endl;
        return 1;
    }

    std::vector<double> X;
    std::vector<double> y;
    double value;

    while (inputFile >> value) {
        X.push_back(value);
        inputFile >> value;
        y.push_back(value);
    }

    inputFile.close();

    int epochs = 1000;
    double learningRate = 0.01;
    LinearRegressionResult result = trainModelCUDA(X, y, epochs, learningRate);

    std::ofstream outputFile("output.txt");
    if (!outputFile.is_open()) {
        std::cerr << "Error al abrir el archivo de salida." << std::endl;
        return 1;
    }

    outputFile << "Pendiente (m): " << result.slope << std::endl;
    outputFile << "Intersección (b): " << result.intercept << std::endl;

    outputFile.close();

    std::cout << "El modelo se ha guardado en 'output.txt'" << std::endl;

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Tiempo de ejecución: " << elapsed.count() << "s" << std::endl;

    return 0;
}
